
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h> //required for rand() only to initialize sample matrices
#include "hipblas.h"

extern "C" __declspec(dllexport)
void tester(double* x){
    double* y,*dy;
    y = (double*)calloc(2, sizeof(double));
    printf("Hello\n");
    for(int i = 0; i < 2; ++i){
        y[i] = x[i] + 2.;
    }
    hipMalloc(&dy, 2 * sizeof(double));
    hipMemcpy(dy,y,2*sizeof(double),hipMemcpyHostToDevice);

    hipMemcpy(x,dy,2 * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(dy);
    free(y);
}
extern "C" __declspec(dllexport)
void mmult(double* A, double* B, double* C, int* m, int* k, int* n){
#ifdef TESTING
    printf("entered dll function mmult\n");
#endif
    int M = *m;
    int K = *k;
    int N = *n;
    double *da,*db,*dc;
    hipMalloc(&da, M * K * sizeof(double));
    hipMalloc(&db, K * N * sizeof(double));
    hipMalloc(&dc, M * N * sizeof(double));

    hipMemcpy(da, A, M * K * sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(db, B, K * N * sizeof(double),hipMemcpyHostToDevice);

#ifdef TESTING
    for(int i = 0; i < M; ++i){
        for(int j = 0; j < K; ++j){
            printf("%7.4f ",A[j * M + i]);
        }
        printf("\n");
    }
    for(int i = 0; i < K; ++i){
        for(int j = 0; j < N; ++j){
            printf("%7.4f ",B[j * K + i]);
        }
        printf("\n");
    }
#endif


    double alpha = 1., beta = 0.;

    hipblasHandle_t han;

    hipblasCreate(&han);
    {//dispensible
        int cublasversion;    
        cublasGetVersion(han, &cublasversion);
        printf("cublas version is %i\n", cublasversion);
    }
        hipblasDgemm(han, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                    M,N,K, 
                    &alpha, 
                    da, M, 
                    db, K, 
                    &beta, 
                    dc, M);

    hipMemcpy(C,dc,M * N * sizeof(double),hipMemcpyDeviceToHost);

#ifdef TESTING
    for(int i = 0; i < M; ++i){
        for(int j = 0; j < N; ++j){
            printf("%7.4f ",C[j * M + i]);
        }
        printf("\n");
    }
#endif
    hipblasDestroy(han);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
}
int main(){
    int M,N,K;
    M = N = K = 4;
    double* A = (double*)calloc(M * K, sizeof(double));
    double* B = (double*)calloc(K * N, sizeof(double));
    double* C = (double*)calloc(M * N, sizeof(double));
    
    //initialize a
    for(int i = 0; i < M * K; ++i){ A[i] = i + 1.;}
    printf("A:\n");
    for(int i = 0; i < M; ++i){
        for(int j = 0; j < K; ++j){
            printf("%7.4f ",A[j * M + i]);
        }
        printf("\n");
    }
    for(int i = 0; i < K * N; ++i){ B[i] = i * 3 - 2.;}
    printf("B:\n");
    for(int i = 0; i < K; ++i){
        for(int j = 0; j < N; ++j){
            printf("%7.4f ",B[j * K + i]);
        }
        printf("\n");
    }

    mmult(A, B, C, &M, &K, &N);
    printf("C:\n");
    for(int i = 0; i < M; ++i){
        for(int j = 0; j < N; ++j){
            printf("%7.4f ",C[j * M + i]);
        }
        printf("\n");
    }

    free(A);
    free(B);
    free(C);
}

